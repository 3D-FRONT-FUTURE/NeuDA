#include "hip/hip_runtime.h"
#include "render.hpp"
#include <iostream>
#include <stdio.h>

__constant__ int height;
__constant__ int width;
__constant__ int num_primitives;
__constant__ float cx, cy, fx, fy;
__constant__ glm::mat3 rotation;
__constant__ glm::vec3 translation;
__constant__ int render_primitives;
#define MIN(a,b) (((a)<(b))?(a):(b))
#define MAX(a,b) (((a)>(b))?(a):(b))

__host__ __device__ static
double calculateSignedArea2(const glm::dvec3& a, const glm::dvec3& b, const glm::dvec3& c) {
    return ((c.x - a.x) * (b.y - a.y) - (b.x - a.x) * (c.y - a.y));
}

__host__ __device__ static
glm::dvec3 calculateBarycentricCoordinate(const glm::dvec3& a, const glm::dvec3& b, const glm::dvec3& c, const glm::dvec3& p) {
    double beta_tri = calculateSignedArea2(a, p, c);
    double gamma_tri = calculateSignedArea2(a, b, p);
    double tri_inv = 1.0f / calculateSignedArea2(a, b, c);
    double beta = beta_tri * tri_inv;
    double gamma = gamma_tri * tri_inv;
    double alpha = 1.0 - beta - gamma;
    return glm::vec3(alpha, beta, gamma);
}

__host__ __device__ static
bool isBarycentricCoordInBounds(const glm::dvec3 barycentricCoord) {
    return barycentricCoord.x >= 0.0 && barycentricCoord.x <= 1.0 &&
           barycentricCoord.y >= 0.0 && barycentricCoord.y <= 1.0 &&
           barycentricCoord.z >= 0.0 && barycentricCoord.z <= 1.0;
}

__host__ __device__ static
double getZAtCoordinate(const glm::dvec3 barycentricCoord, const glm::dvec3& a, const glm::dvec3& b, const glm::dvec3& c) {
    return (barycentricCoord.x * a.z
        + barycentricCoord.y * b.z
        + barycentricCoord.z * c.z);
}

__device__ int CompactRGBToInt(const glm::vec3& rgb) {
    return ((int)(rgb.x * 255) << 16)
    + ((int)(rgb.y * 255) << 8)
    + ((int)(rgb.z * 255))
    + (255 << 24);
}

__device__ void atomicExchRGBZ(int* zbuffer, int* image, int z, int rgb) {
    while (true) {
        int expected_rgb = *image;
        int expected_z = *zbuffer;
        if (expected_z > z)
            break;

        int old_rgb = atomicCAS(image, expected_rgb, rgb);

        if (old_rgb == expected_rgb)
            break;
    }
}

__global__ void NaiveRender_gpu(int* color) {
    int pixel = blockIdx.x * blockDim.x + threadIdx.x;
    if (pixel >= height * width)
        return;
    int y = pixel / width;
    int x = pixel - width * y;

    color[pixel] = CompactRGBToInt(glm::vec3(y / (float)height, x / (float)width, 1));
}

__global__ void Render_gpu(glm::vec3* positions, glm::ivec3* indices, int* color, int* findices, int* zbuffer) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= num_primitives)
        return;

    glm::ivec3 face = indices[idx];
    glm::dvec3 p1 = glm::dvec3(rotation * positions[face[0]] + translation);
    glm::dvec3 p2 = glm::dvec3(rotation * positions[face[1]] + translation);
    glm::dvec3 p3 = glm::dvec3(rotation * positions[face[2]] + translation);

    //opengl coordinator to opencv coordinator
    p1.y *= -1;
    p1.z *= -1;
    p2.y *= -1;
    p2.z *= -1;
    p3.y *= -1;
    p3.z *= -1;

    glm::dvec3 n = glm::cross(p2 - p1, p3 - p1);
    if (p1.z < 0.02 || p2.z < 0.02 || p3.z < 0.02 || n.z > 0)
        return;

    p1.z = 1.0f / p1.z;
    p2.z = 1.0f / p2.z;
    p3.z = 1.0f / p3.z;

    p1.x = p1.x * p1.z;
    p1.y = p1.y * p1.z;
    p2.x = p2.x * p2.z;
    p2.y = p2.y * p2.z;
    p3.x = p3.x * p3.z;
    p3.y = p3.y * p3.z;

    int minX = (MIN(p1.x, MIN(p2.x, p3.x)) * fx + cx);
    int minY = (MIN(p1.y, MIN(p2.y, p3.y)) * fy + cy);
    int maxX = (MAX(p1.x, MAX(p2.x, p3.x)) * fx + cx) + 0.999999f;
    int maxY = (MAX(p1.y, MAX(p2.y, p3.y)) * fy + cy) + 0.999999f;

    minX = MAX(0, minX);
    minY = MAX(0, minY);
    maxX = MIN(width, maxX);
    maxY = MIN(height, maxY);

    for (int py = minY; py <= maxY; ++py) {
        for (int px = minX; px <= maxX; ++px) {
            if (px < 0 || px >= width || py < 0 || py >= height)
                continue;

            float x = (px - cx) / fx;
            float y = (py - cy) / fy;

            glm::dvec3 baryCentricCoordinate = calculateBarycentricCoordinate(p1, p2, p3, glm::dvec3(x, y, 0));
            if (isBarycentricCoordInBounds(baryCentricCoordinate)) {
                int pixel = py * width + px;

                float z = getZAtCoordinate(baryCentricCoordinate, p1, p2, p3);
                int z_quantize = z * 100000;

                int original_z = atomicMax(&zbuffer[pixel], z_quantize);

                if (original_z < z_quantize) {
                    glm::vec3 rgb = baryCentricCoordinate;
                    if (render_primitives == 0) {
                        atomicExchRGBZ(&zbuffer[pixel], &color[pixel], z_quantize, CompactRGBToInt(rgb));
                    } else {
                        atomicExchRGBZ(&zbuffer[pixel], &findices[pixel], z_quantize, idx);
                    }
                }
            }
        }
    }
}

__global__ void FetchDepth_gpu(int* z, float* depth) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= height * width)
        return;
    int px = idx % width;
    int py = idx / width;
    int z_value = z[py * width + px];
    float d = 0;
    if (z_value > 0)
        d = 100000.0 / z_value;
    depth[py * width + px] = d;
}

__global__ void FetchVMap_gpu(int* d_z, int* findices, glm::vec3* positions, glm::ivec3* faces, glm::ivec3* vindices, glm::vec3* vweights) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= height * width)
        return;
    if (d_z[idx] == 0) {
        vindices[idx] = glm::ivec3(0, 0, 0);
        vweights[idx] = glm::vec3(0, 0, 0);
        return;	
    }
    int px = idx % width;
    int py = idx / width;

    glm::ivec3 face = faces[findices[idx]];
    vindices[idx] = face;
    glm::dvec3 p1 = glm::dvec3(rotation * positions[face[0]] + translation);
    glm::dvec3 p2 = glm::dvec3(rotation * positions[face[1]] + translation);
    glm::dvec3 p3 = glm::dvec3(rotation * positions[face[2]] + translation);

    // opengl coordinator to opencv coordinator
    p1.y *= -1;
    p1.z *= -1;
    p2.y *= -1;
    p2.z *= -1;
    p3.y *= -1;
    p3.z *= -1;

    if (p1.z < 0.2 || p2.z < 0.2 || p3.z < 0.2) {
        vindices[idx] = glm::ivec3(0, 0, 0);
        vweights[idx] = glm::vec3(0, 0, 0);
        return;
    }

    p1.z = 1.0f / p1.z;
    p2.z = 1.0f / p2.z;
    p3.z = 1.0f / p3.z;

    p1.x = p1.x * p1.z;
    p1.y = p1.y * p1.z;
    p2.x = p2.x * p2.z;
    p2.y = p2.y * p2.z;
    p3.x = p3.x * p3.z;
    p3.y = p3.y * p3.z;


    glm::dvec3 barycentric = calculateBarycentricCoordinate(p1, p2, p3, glm::dvec3((px - cx) / fx, (py - cy) / fy, 0));
    double inv_z = 1.0f / getZAtCoordinate(barycentric, p1, p2, p3);
    
    vweights[idx] = glm::vec3(barycentric.x * p1.z * inv_z, barycentric.y * p2.z * inv_z, barycentric.z * p3.z * inv_z);
}

void NaiveRender(FrameBuffer& frameBuffer) {
    int num_pixels = frameBuffer.row * frameBuffer.col;
    hipMemcpyToSymbol(HIP_SYMBOL(height), &frameBuffer.row, sizeof(int), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(width), &frameBuffer.col, sizeof(int), 0, hipMemcpyHostToDevice);

    NaiveRender_gpu<<<(num_pixels + 255) / 256, 256>>>(frameBuffer.d_colors);
}

void Render(VertexBuffer& vertexBuffer, FrameBuffer& frameBuffer, int renderPrimitive) {
    hipMemcpyToSymbol(HIP_SYMBOL(height), &frameBuffer.row, sizeof(int), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(width), &frameBuffer.col, sizeof(int), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(cx), &frameBuffer.cx, sizeof(float), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(cy), &frameBuffer.cy, sizeof(float), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(fx), &frameBuffer.fx, sizeof(float), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(fy), &frameBuffer.fy, sizeof(float), 0, hipMemcpyHostToDevice);

    hipMemcpyToSymbol(HIP_SYMBOL(render_primitives), &renderPrimitive, sizeof(int), 0, hipMemcpyHostToDevice);

    hipMemcpyToSymbol(HIP_SYMBOL(rotation), &vertexBuffer.rotation, sizeof(float) * 9, 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(translation), &vertexBuffer.translation, sizeof(float) * 3, 0, hipMemcpyHostToDevice);

    hipMemcpyToSymbol(HIP_SYMBOL(num_primitives), &vertexBuffer.num_indices, sizeof(int), 0, hipMemcpyHostToDevice);

    Render_gpu<<<(vertexBuffer.num_indices + 255) / 256, 256>>>(vertexBuffer.d_positions, vertexBuffer.d_indices, frameBuffer.d_colors, frameBuffer.d_findices, frameBuffer.d_z);
}

void FetchDepth(FrameBuffer& frameBuffer) {
    int num_pixels = frameBuffer.row * frameBuffer.col;
    hipMemcpyToSymbol(HIP_SYMBOL(height), &frameBuffer.row, sizeof(int), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(width), &frameBuffer.col, sizeof(int), 0, hipMemcpyHostToDevice);
    FetchDepth_gpu<<<(num_pixels+255)/256, 256>>>(frameBuffer.d_z, frameBuffer.d_depth);	
}

void FetchVMap(VertexBuffer& vertexBuffer, FrameBuffer& frameBuffer) {
    int num_pixels = frameBuffer.row * frameBuffer.col;
    hipMemcpyToSymbol(HIP_SYMBOL(height), &frameBuffer.row, sizeof(int), 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(width), &frameBuffer.col, sizeof(int), 0, hipMemcpyHostToDevice);
    FetchVMap_gpu<<<(num_pixels+255)/256, 256>>>(frameBuffer.d_z, frameBuffer.d_findices, vertexBuffer.d_positions, vertexBuffer.d_indices, frameBuffer.d_vindices, frameBuffer.d_vweights);
}

__global__ void Rotate_gpu(glm::vec3* output, glm::vec3* input, glm::mat3 rot, int count) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= count)
        return;
    output[idx] = rot * input[idx];
}

void rotate_cuda(glm::vec3* output, glm::vec3* input, glm::mat3 rot, int count) {
    Rotate_gpu<<<(count + 255) / 256, 256>>>(output, input, rot, count);
}
